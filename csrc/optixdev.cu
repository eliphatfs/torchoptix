#include "hip/hip_runtime.h"
#include <cstdint>
#include <cfloat>
#include "optixinc.h"
#include "hip/hip_vector_types.h"
#include <optix_device.h>

#ifdef __INTELLISENSE__
int __float_as_int(float in);
float __int_as_float(int in);
// Add other intrinsics as needed
#endif

extern "C" {
    __constant__ LaunchParams optixLaunchParams;

    __global__ void __raygen__rg()
    {
        const uint3 launch_index = optixGetLaunchIndex();

        // Load ray origin and direction from some buffer
        float3 ray_origin = ((float3*)optixLaunchParams.rays_o)[launch_index.x];
        float3 ray_direction = ((float3*)optixLaunchParams.rays_d)[launch_index.x];

        // Trace the ray
        uint32_t i, t_as_int;
        optixTrace(
            optixLaunchParams.traversable,
            ray_origin,
            ray_direction,
            0.0f,
            optixLaunchParams.t_max,
            0.0f,
            OptixVisibilityMask(255),
            OPTIX_RAY_FLAG_DISABLE_ANYHIT,  // OPTIX_RAY_FLAG_NONE,
            0,             // SBT offset
            1,             // SBT stride
            0,             // missSBTIndex 
            i, t_as_int
        );

        // Obtain hit information (like triangle ID, hit point)
        // Process the hit information
        float t = __int_as_float(t_as_int);
        ((uint32_t*)optixLaunchParams.out_i)[launch_index.x] = i;
        ((float*)optixLaunchParams.out_t)[launch_index.x] = t;
    }

    __global__ void __closesthit__ch()
    {
        optixSetPayload_0(optixGetPrimitiveIndex());
        optixSetPayload_1(__float_as_int(optixGetRayTmax()));
    }

    __global__ void __miss__far()
    {
        optixSetPayload_0(0);
        optixSetPayload_1(__float_as_int(optixLaunchParams.t_max));
    }
}
